#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <iterator>
#include <cmath>
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <stdio.h>  // Pour fprintf et stderr
#include <stdlib.h> // Pour les fonctions standard C comme malloc
#include <stdint.h>
#include "picojson.h"
// Définition de l'énumération pour le type de fractale
enum Type_Fractal
{
    Mandelbrot,
    Julia
};
bool DEBUG = false;

// Définition de la structure Complex pour représenter les nombres complexes
struct Complex
{
    double x, y; // Partie réelle et imaginaire

    // Constructeur pour initialiser un nombre complexe
    __host__ __device__
    Complex(double a = 0.0, double b = 0.0) : x(a), y(b) {}

    // Surcharge de l'opérateur + pour l'addition de deux nombres complexes
    __host__ __device__
        Complex
        operator+(const Complex &other) const
    {
        return Complex(x + other.x, y + other.y);
    }

    // Surcharge de l'opérateur - pour la soustraction de deux nombres complexes
    __host__ __device__
        Complex
        operator-(const Complex &other) const
    {
        return Complex(x - other.x, y - other.y);
    }

    // Surcharge de l'opérateur * pour la multiplication de deux nombres complexes
    __host__ __device__
        Complex
        operator*(const Complex &other) const
    {
        return Complex(x * other.x - y * other.y, x * other.y + y * other.x);
    }

    // Fonction pour calculer la norme d'un nombre complexe
    __host__ __device__ double norm() const
    {
        return sqrt(x * x + y * y);
    }

    // Fonction pour élever un nombre complexe à une puissance donnée
    __host__ __device__
        Complex
        power(double p) const
    {
        double radius = sqrt(x * x + y * y);
        double angle = atan2(y, x);
        double radius_p = pow(radius, p);
        double angle_p = p * angle;

        return Complex(radius_p * cos(angle_p), radius_p * sin(angle_p));
    }
};

// Définition de la structure ParameterPicture pour stocker les paramètres de l'image fractale
__host__ __device__ struct ParameterPicture
{
    long len_image_per_tuile; // Longueur globale en 3D
    long nb_tuiles;           // Longueur locale en 2D
    long no_tuile;
    double2 start;             // Point de départ de l'image
    double size;               // Taille d'un côté de l'image
    Type_Fractal type_fractal; // Type de fractale (Mandelbrot ou Julia)
    double2 coef_julia;        // Coefficients pour la fractale de Julia
    double power_value;        // Valeur de la puissance
    long iter_max;             // Nombre maximal d'itérations
    long id;                   // Identifiant de l'image
    double pas_tuile;

    // Constructeur pour initialiser un objet ParameterPicture
    __host__ __device__ ParameterPicture(long no_tuile, long len_image_per_tuile, long nb_tuiles, double2 start, double size, double power_value, long iter_max, Type_Fractal type_fractal, double2 coef_julia = make_double2(0.0, 0.0))
        : no_tuile(no_tuile), power_value(power_value), iter_max(iter_max), type_fractal(type_fractal), coef_julia(coef_julia), len_image_per_tuile(len_image_per_tuile), nb_tuiles(nb_tuiles), start(start), size(size), pas_tuile(size / ((double)(nb_tuiles))) {};

    __host__ __device__ size_t get_size_array_2D_tuile() const
    {
        return (size_t)len_image_per_tuile * (size_t)len_image_per_tuile;
    }

    __host__ __device__ size_t get_len_global_image() const
    {
        return (size_t)nb_tuiles * (size_t)len_image_per_tuile;
    }

    __host__ __device__ long2 get_x_y_tuile_no_from_no_tuile() const
    {
        long x = no_tuile % nb_tuiles;
        long y = (no_tuile - x) / nb_tuiles;

        return make_long2(x, y);
    }

    __host__ __device__ long2 get_x_y_tuile_px_from_no_tuile() const
    {
        long2 pose_no = get_x_y_tuile_no_from_no_tuile();

        return make_long2(pose_no.x * len_image_per_tuile, pose_no.y * len_image_per_tuile);
    }

    __host__ __device__ double2 get_x_y_tuile_double_from_no_tuile() const
    {
        long2 pose_no = get_x_y_tuile_no_from_no_tuile();

        return make_double2((double)pose_no.x * pas_tuile, (double)pose_no.y * pas_tuile);
    }

    __host__ __device__ double2 get_x_y_globale_double_from_x_y(int x, int y) const
    {
        double2 pos_po_double = get_x_y_tuile_double_from_no_tuile();

        return make_double2(start.x + pos_po_double.x + (double)x / ((double)(len_image_per_tuile - 1)) * pas_tuile, start.y + pos_po_double.y + (double)y / ((double)(len_image_per_tuile - 1)) * pas_tuile);
    }

    __host__ __device__ long get_index_long_from_x_y(int x, int y) const
    {
        if (x < 0 || x >= (len_image_per_tuile))
        {
            return -2;
        }

        if (y < 0 || y >= (len_image_per_tuile))
        {
            return -1;
        }

        return (long)x + (long)y * len_image_per_tuile;
    }

    // Fonction pour imprimer les paramètres de l'image dans un fichier
    __host__ void print_file(std::string path_file) const
    {
        std::ofstream myfile;
        myfile.open(path_file, std::ios::app);
        myfile << "no_tuile = " << no_tuile << std::endl;

        myfile << "len_image_per_tuile = " << len_image_per_tuile << std::endl;
        myfile << "nb_tuiles = " << nb_tuiles << std::endl;
        myfile << "pas_tuile= " << pas_tuile << std::endl;

        myfile << "start_x = " << start.x << std::endl;
        myfile << "start_y = " << start.y << std::endl;

        myfile << "size = " << size << std::endl;
        myfile << "type_fractal = " << type_fractal << std::endl;
        myfile << "coef_julia_x = " << coef_julia.x << std::endl;
        myfile << "coef_julia_y = " << coef_julia.y << std::endl;

        myfile << "power_value = " << power_value << std::endl;
        myfile << "iter_max = " << iter_max << std::endl;

        myfile.close();
    }
};

__host__ __device__ int comptute_fractale(ParameterPicture parameter_picture, int x, int y)
{

    // Obtenir la position complexe correspondante
    double2 pos_double = parameter_picture.get_x_y_globale_double_from_x_y(x, y);
    Complex z(pos_double.x, pos_double.y);
    Complex c(pos_double.x, pos_double.y);

    // Si le type de fractale est Julia, utiliser les coefficients de Julia
    if (parameter_picture.type_fractal == Type_Fractal::Julia)
    {
        c.x = parameter_picture.coef_julia.x;
        c.y = parameter_picture.coef_julia.y;
    }

    int iter = 0;

    // Calculer le nombre d'itérations pour la fractale
    while (z.norm() < 2.0 && iter < parameter_picture.iter_max)
    {
        z = z.power(parameter_picture.power_value) + c;
        iter++;
    }

    return iter;
}

// Kernel CUDA pour générer une image fractale
__global__ void Kernel_Picture(ParameterPicture parameter_picture, unsigned char *data)
{
    // Calcul des indices 3D pour chaque thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Obtenir l'index 2D correspondant
    long index = parameter_picture.get_index_long_from_x_y(idx, idy);

    // Si l'index est valide
    if (index >= 0)
    {
        int iter = comptute_fractale(parameter_picture, idx, idy);

        data[index] = (unsigned char)(iter % 256);
    }
}

void C_Picture(ParameterPicture parameter_picture)
{

    parameter_picture.print_file("file.txt");
    for (int idx = 0; idx < parameter_picture.len_image_per_tuile; idx++)
    {
        for (int idy = 0; idy < parameter_picture.len_image_per_tuile; idy++)
        {
            double2 pos_double = parameter_picture.get_x_y_globale_double_from_x_y(idx, idy);
            long index = parameter_picture.get_index_long_from_x_y(idx, idy);
            int iter = comptute_fractale(parameter_picture, idx, idy);

            std::cout << index << " ==> " << iter << " ==> " << idx << "/" << idy << " ==> " << pos_double.x << "/" << pos_double.y << std::endl;
        }
    }
}
// Fonction pour exécuter le kernel CUDA
extern "C" hipError_t RUN(long no_tuile, long nb_tuiles, unsigned char *datas)
{

    std::string json_file = "config.json";

    std::ifstream t(json_file);
    std::string json_data((std::istreambuf_iterator<char>(t)),
                          std::istreambuf_iterator<char>());

    picojson::value v;

    picojson::parse(v, json_data);
    int id_cuda = std::stoi(v.get("id_cuda").to_str());
    // long no_tuile = std::stol(v.get("no_tuile").to_str());
    long len_image_per_tuile = std::stol(v.get("len_image_per_tuile").to_str());
    // long nb_tuiles = std::stol(v.get("nb_tuiles").to_str());
    double2 start = make_double2(std::stod(v.get("start_x").to_str()), std::stod(v.get("start_y").to_str()));
    double size_double = std::stod(v.get("size").to_str());
    double power_value = std::stod(v.get("power_value").to_str());
    long iter_max = std::stol(v.get("iter_max").to_str());
    Type_Fractal type_fractal = Type_Fractal::Mandelbrot;
    // long no_tuile = std::stol(v.get("no_tuile").to_str());
    ParameterPicture parameter_picture(no_tuile, len_image_per_tuile, nb_tuiles, start, size_double, power_value, iter_max, type_fractal);
    // ParameterPicture parameter_picture() Type_Fractal::Mandelbrot);

    // Calculer la taille des données à allouer
    size_t size = parameter_picture.get_size_array_2D_tuile() * sizeof(unsigned char);
    unsigned char *dev_datas = 0;
    hipError_t cudaStatus;

    // Définir la configuration des threads et des blocs
    const dim3 threadsPerBlock(16, 16, 1);
    const dim3 numBlocks((parameter_picture.len_image_per_tuile + threadsPerBlock.x - 1) / threadsPerBlock.x,
                         (parameter_picture.len_image_per_tuile + threadsPerBlock.y - 1) / threadsPerBlock.y,
                         1);

    // Sélectionner le GPU à utiliser
    cudaStatus = hipSetDevice(id_cuda);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allouer de la mémoire sur le GPU pour les données
    cudaStatus = hipMalloc((void **)&dev_datas, size);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Lancer le kernel CUDA
    if (DEBUG)
        std::cout << "Start Kernel_Picture" << std::endl;
    Kernel_Picture<<<numBlocks, threadsPerBlock>>>(parameter_picture, dev_datas);
    if (DEBUG)
        std::cout << "End Kernel_Picture" << std::endl;

    if (DEBUG)
        std::cout << "Start Vérifier si le lancement du kernel a échoué" << std::endl;
    // Vérifier si le lancement du kernel a échoué
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Kernel_Picture launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    if (DEBUG)
        std::cout << "End  Vérifier si le lancement du kernel a échoué" << std::endl;

    if (DEBUG)
        std::cout << "Start Attendre la fin de l'exécution du kernel" << std::endl;
    // Attendre la fin de l'exécution du kernel
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel_Picture!\n", cudaStatus);
        goto Error;
    }
    if (DEBUG)
        std::cout << "End Attendre la fin de l'exécution du kernel" << std::endl;

    // Copier les données du GPU vers la mémoire de l'hôte
    if (DEBUG)
        std::cout << "Start Copier les données du GPU vers la mémoire de l'hôte" << std::endl;
    cudaStatus = hipMemcpy(datas, dev_datas, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    if (DEBUG)
        std::cout << "End Copier les données du GPU vers la mémoire de l'hôte" << std::endl;

    // Libérer la mémoire allouée sur le GPU
    if (DEBUG)
        std::cout << "Start Libérer la mémoire allouée sur le GPU" << std::endl;
    hipFree(dev_datas);
    if (DEBUG)
        std::cout << "End Libérer la mémoire allouée sur le GPU" << std::endl;

    // Réinitialiser le GPU
    if (DEBUG)
        std::cout << "Start Réinitialiser le GPU" << std::endl;
    cudaStatus = hipDeviceReset();
    if (DEBUG)
        std::cout << "End Réinitialiser le GPU" << std::endl;
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return cudaStatus;
    }

    return hipSuccess;

Error:
    // En cas d'erreur, libérer la mémoire allouée sur le GPU
    hipFree(dev_datas);
    return cudaStatus;
}

int main()
{
    std::string json_file = "config.json";

    std::ifstream t(json_file);
    std::string json_data((std::istreambuf_iterator<char>(t)),
                          std::istreambuf_iterator<char>());

    picojson::value v;

    picojson::parse(v, json_data);
    int id_cuda = std::stoi(v.get("id_cuda").to_str());
    long no_tuile = std::stol(v.get("no_tuile").to_str());
    long len_image_per_tuile = std::stol(v.get("len_image_per_tuile").to_str());
    long nb_tuiles = std::stol(v.get("nb_tuiles").to_str());
    double2 start = make_double2(std::stod(v.get("start_x").to_str()), std::stod(v.get("start_y").to_str()));
    double size_double = std::stod(v.get("size").to_str());
    double power_value = std::stod(v.get("power_value").to_str());
    long iter_max = std::stol(v.get("iter_max").to_str());
    Type_Fractal type_fractal = Type_Fractal::Mandelbrot;
    // long no_tuile = std::stol(v.get("no_tuile").to_str());
    ParameterPicture parameter_picture(no_tuile, len_image_per_tuile, nb_tuiles, start, size_double, power_value, iter_max, type_fractal);

    C_Picture(parameter_picture);

    unsigned char *datas_G = 0;
    size_t size = 1024 * 1024 * sizeof(unsigned char);
    std::cout << "début malloc datas_BW" << std::endl;
    datas_G = (unsigned char *)malloc(size);
    std::cout << "fin malloc datas_BW" << std::endl;

    hipError_t cudaStatus;

    std::cout << "début du RUN Cuda" << std::endl;
    cudaStatus = RUN(0, 2, datas_G);
    std::cout << "fin du RUN Cuda" << std::endl;
}